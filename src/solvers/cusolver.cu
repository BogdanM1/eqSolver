#include "cusolver.h"
#include "solverOptions.h"

void CuSolver::solve(equationSystem& sistem)
{  

	int*d_irows,*d_jcols;
	double*d_vals,*d_vekt;
    int sing=0;

	hipsolverSpHandle_t cusolverH;
	csrqrInfo_t info;             
	hipsparseMatDescr_t descrA;
	hipsparseHandle_t handle;		
	
	hipMalloc((void**)&d_irows,sistem.nonzero_count*sizeof(int));
    hipMalloc((void**)&d_jcols,sistem.nonzero_count*sizeof(int));
    hipMalloc((void**)&d_vals,sistem.nonzero_count*sizeof(double));
    hipMalloc((void**)&d_vekt,sistem.eq_count*sizeof(double));

	hipMemcpy(d_irows, &sistem.rows[0], (size_t)(sistem.nonzero_count*sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(d_jcols, &sistem.cols[0], (size_t)(sistem.nonzero_count*sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(d_vals,  &sistem.vals[0], (size_t)(sistem.nonzero_count*sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(d_vekt,  &sistem.rhs[0], (size_t)(sistem.eq_count*sizeof(double)), hipMemcpyHostToDevice);

	
	cusolverH=NULL; info=NULL; descrA=NULL; handle=NULL;
	hipsparseCreate(&handle);
	hipsolverSpCreate(&cusolverH);
	hipsparseCreateMatDescr(&descrA);
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	if(solverOptions::get().isym==1)
	{
		hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
		if(solverOptions::get().cusolver.filltype == "lower")
			hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_LOWER);
		else 
			hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_UPPER);
	}
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);
	cusolverSpCreateCsrqrInfo(&info);
	hipsparseXcoo2csr(handle,d_irows,sistem.nonzero_count,sistem.eq_count, d_irows,HIPSPARSE_INDEX_BASE_ONE);
 			
	hipsolverSpDcsrlsvchol(cusolverH, sistem.eq_count, sistem.nonzero_count, descrA, 
	                     d_vals, d_irows, d_jcols, d_vekt, 
						 solverOptions::get().cusolver.tolerance, solverOptions::get().cusolver.reordering, d_vekt, &sing);
  
	hipMemcpy(&sistem.rhs[0], d_vekt, sizeof(double)*sistem.eq_count ,hipMemcpyDeviceToHost);
		
	hipFree(d_irows);     
    hipFree(d_jcols);
    hipFree(d_vals);
    hipFree(d_vekt); 
	hipsparseDestroy(handle);
	hipsparseDestroyMatDescr(descrA);
	cusolverSpDestroyCsrqrInfo(info);
	hipsolverSpDestroy(cusolverH);		
	
}


